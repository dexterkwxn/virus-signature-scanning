#include "hip/hip_runtime.h"
/*
        CS3210 Assignment 2
        CUDA Virus Scanning

        Most of your CUDA code should go in here.

        Feel free to change any code in the skeleton, as long as you conform
        to the input and output formats specified in the assignment pdf.

        If you rename this file or add new files, remember to modify the
        Makefile! Just make sure (haha) that the default target still builds
        your program, and you don't rename the program (`scanner`).

        The skeleton demonstrates how asnychronous kernel launches can be
        done; it is up to you to decide (and implement!) the parallelisation
        paradigm for the kernel. The provided implementation is not great,
        since it launches one kernel per file+signature combination (a lot!).
        You should try to do more work per kernel in your implementation.

        You can launch as many kernels as you want; if any preprocessing is
        needed for your algorithm of choice, you can also do that on the GPU
        by running different kernels.

        'defs.h' contains the definitions of the structs containing the input
        and signature data parsed by the provided skeleton code; there should
        be no need to change it, but you can if you want to.

        'common.cpp' contains the aforementioned parsing for the input files.
        The input files are already efficiently read with mmap(), so there
        should be little to no gain trying to optimise that portion of the
        skeleton.

        Remember: print any debugging statements to STDERR!
*/

#include <vector>

#include "defs.h"

constexpr int N = 64;

//__device__ int global_counter = 0;

__global__ void matchFile(const char *file_name, const uint8_t *file_data,
                          size_t file_len, char **sig_names,
                          const char *sigs_buf, int32_t *sig_offsets) {

  int block_offset =
      gridDim.x * gridDim.y * blockIdx.z + gridDim.x * blockIdx.y + blockIdx.x;

  int thread_offset = blockDim.x * blockDim.y * threadIdx.z +
                      blockDim.x * threadIdx.y + threadIdx.x;

  int len = sig_offsets[block_offset + 1] - sig_offsets[block_offset];
  const char *signature = sigs_buf + sig_offsets[block_offset];

  int file_blk_sz = (file_len + N - 1) / N;
  int start_idx = thread_offset * file_blk_sz;
  int end_idx = (thread_offset + 1) * file_blk_sz;
  for (size_t i = start_idx; i < end_idx; ++i) {
    if (i >= file_len) {
      // exceed file size, stop
      return;
    }
    bool match = true;
    int file_data1, file_data2;
    int first_value, second_value;
    for (int j = 0; j < len / 2; ++j) {

      if (!match)
        break;
      if (i + 2 * j + 1 >= file_len) {
        match = 0;
        break;
      }
      // convert byte that we are comparing
      char first_half = signature[2 * j];
      char second_half = signature[2 * j + 1];

      file_data1 = file_data[i + j] / 16;
      file_data2 = file_data[i + j] % 16;

      if (first_half != '?') {
        if (first_half > 96) {
          first_value = first_half - 'a' + 10;
        } else if (first_half > 47) {
          first_value = first_half - '0';
        }

        match = match && (file_data1 == first_value);
        // if (match) printf("file_data1: %d, first_value: %d, i: %ld, j: %d\n",
        // file_data1, first_value, i, j);
      }

      if (second_half != '?') {
        if (second_half > 96) {
          second_value = second_half - 'a' + 10;
        } else if (second_half > 47) {
          second_value = second_half - '0';
        }

        match = match && (file_data2 == second_value);
        // if (match) printf("file_data2: %d, second_value: %d, i: %ld, j:
        // %d\n", file_data2, second_value, i, j);
      }
    }
    // if match, store it
    if (match) {
      //      printf("-------------------------MATCH----------------------- i:
      //      %ld, "
      //             "sig_idx: %d \n",
      //             i, block_offset);
      printf("%s: %s\n", file_name, sig_names[block_offset]);
    }
  }

  __syncthreads();
  // we assume the same substring cannot represent 2 viruses
}

void runScanner(std::vector<Signature> &signatures,
                std::vector<InputFile> &inputs) {
  {
    hipDeviceProp_t prop;
    check_cuda_error(hipGetDeviceProperties(&prop, 0));

    fprintf(stderr, "cuda stats:\n");
    fprintf(stderr, "  # of SMs: %d\n", prop.multiProcessorCount);
    fprintf(stderr, "  global memory: %.2f MB\n",
            prop.totalGlobalMem / 1024.0 / 1024.0);
    fprintf(stderr, "  shared mem per block: %zu bytes\n",
            prop.sharedMemPerBlock);
    fprintf(stderr, "  constant memory: %zu bytes\n", prop.totalConstMem);
  }

  std::vector<hipStream_t> streams{};
  streams.resize(inputs.size());

  std::vector<uint8_t *> file_bufs{};

  for (size_t i = 0; i < inputs.size(); i++) {
    hipStreamCreate(&streams[i]);

    // allocate memory on the device for the file
    uint8_t *ptr = 0;
    check_cuda_error(hipMalloc(&ptr, inputs[i].size));
    file_bufs.push_back(ptr);
  }

  // allocate memory for the signatures
  char *sigs_buf;
  int32_t *sig_offsets;
  int32_t cur_offset = 0;
  char **sig_names;
  check_cuda_error(hipMallocManaged(&sig_offsets, (signatures.size() + 1) *
                                                       sizeof(int32_t)));
  check_cuda_error(hipMallocManaged(&sig_names, signatures.size()));
  for (size_t i = 0; i < signatures.size(); i++) {
    cur_offset += signatures[i].size;
  }
  check_cuda_error(hipMalloc(&sigs_buf, cur_offset));
  sig_offsets[signatures.size()] = cur_offset;
  cur_offset = 0;
  for (size_t i = 0; i < signatures.size(); i++) {
    sig_offsets[i] = cur_offset;
    hipMemcpy(sigs_buf + cur_offset, signatures[i].data, signatures[i].size,
               hipMemcpyHostToDevice);
    check_cuda_error(hipMalloc(&sig_names[i], signatures[i].name.size()));
    hipMemcpy(sig_names[i], signatures[i].name.data(),
               signatures[i].name.size(), hipMemcpyHostToDevice);
    cur_offset += signatures[i].size;
  }

  std::vector<char *> file_names(inputs.size());
  dim3 blockDimensions(1, 1, N);
  for (size_t file_idx = 0; file_idx < inputs.size(); file_idx++) {
    // asynchronously copy the file contents from host memory
    // (the `inputs`) to device memory (file_bufs, which we allocated above)
    hipMemcpyAsync(file_bufs[file_idx], inputs[file_idx].data,
                    inputs[file_idx].size, hipMemcpyHostToDevice,
                    streams[file_idx]);
    check_cuda_error(
        hipMalloc(&file_names[file_idx], inputs[file_idx].name.size()));
    hipMemcpyAsync(file_names[file_idx], inputs[file_idx].name.data(),
                    inputs[file_idx].name.size(), hipMemcpyHostToDevice,
                    streams[file_idx]);
    // pass in the stream here to do this async

    dim3 gridDimensions(1, 1, signatures.size());
    matchFile<<<gridDimensions, blockDimensions, 0, streams[file_idx]>>>(
        file_names[file_idx], file_bufs[file_idx], inputs[file_idx].size,
        sig_names, sigs_buf, sig_offsets);
  }

  hipFree(sigs_buf);
  hipFree(sig_offsets);

  // free the device memory, though this is not strictly necessary
  // (the CUDA driver will clean up when your program exits)
  for (auto buf : file_bufs)
    hipFree(buf);

  // clean up streams (again, not strictly necessary)
  for (auto &s : streams)
    hipStreamDestroy(s);
}
